#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <inttypes.h>
#include <string.h>
#include <stdlib.h>

#define CUDA_CHECK(cmd) {hipError_t error = cmd; if(error!=hipSuccess){printf("<%s>:%i ",__FILE__,__LINE__); printf("[CUDA] Error: %s\n", hipGetErrorString(error));}}
#define CUDA_CHECK_KERNEL {hipError_t error = hipGetLastError(); if(error!=hipSuccess){printf("<%s>:%i ",__FILE__,__LINE__); printf("[CUDA] Error: %s\n", hipGetErrorString(error));}}


__global__ void count_combinations(int64_t *preset, uint64_t *found_combinations){
  int idx=blockIdx.x*blockDim.x+threadIdx.x;

  int64_t bv = preset[4 * idx];
  int64_t bh = preset[4 * idx + 1];
  int64_t bu = preset[4 * idx + 2];
  int64_t bd = preset[4 * idx + 3];
  
  //TODO Input Algo
  
}



int main(int argc, char ** argv){
    if(argc < 2){
        printf("Not enough arguments\n");
        return 1;
    }
    int32_t N = atoi(argv[1]);
	
	
	int blocksize=1024;
	dim3 dimBlock(blocksize);
	dim3 dimGrid(1);
	
	//TODO
	int size = blocksize;
	
	uint64_t * found_combinations_host = malloc(blocksize,sizeof(uint64_t));
	for(int i = 0; i < blocksize; i++){
		found_combinations_host[i] = 0;
	}
	
	
	int64_t preset_host[blocksize*4];
	
	CUDA_CHECK(hipMalloc((void**)&preset_device,size*4*sizeof(int64_t)));
	CUDA_CHECK(hipMalloc((void**)&found_combinations_device,size*sizeof(uint64_t)));
	
	CUDA_CHECK(hipMemcpy(preset_device,preset_host,size*4*sizeof(int64_t),hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(found_combinations_device,found_combinations_host,size*sizeof(uint64_t),hipMemcpyHostToDevice));
	
	
	//TODO
	count_combinations<<<dimGrid,dimBlock>>>(preset_device,found_combinations_device);
	CUDA_CHECK_KERNEL
	
	
	
	CUDA_CHECK(hipMemcpy(found_combinations_host,found_combinations_device,size*sizeof(uint64_t),hipMemcpyDeviceToHost));
	
	uint_64 found_combinations = 0;
	for(int i = 0; i < size; i++){
		found_combinations += found_combinations_host[i];
	}
	printf("\nFound Combinations: %" PRIu64 "\n", found_combinations);
	
	
	CUDA_CHECK(hipFree(found_combinations_device));
	CUDA_CHECK(hipFree(preset_device));
	
}